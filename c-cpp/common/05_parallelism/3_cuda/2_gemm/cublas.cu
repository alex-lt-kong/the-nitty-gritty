// Originally
// https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuBLAS/Level-3/gemm
// Doc: https://docs.nvidia.com/cuda/cublas/#cublas-t-gemm
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../../utils.h"
#include "../../utils.hpp"
#include "../cublas_utils.h"

using dtype = float;

int main(int argc, char *argv[]) {
  int device;
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDevice(&device));
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));
  std::cout << "GPU: " << prop.name << std::endl;
  hipblasHandle_t cublasH = NULL;

  size_t m = 30000;
  size_t k = 8000;
  size_t n = 11000;
  const size_t lda = m; // ld means "leading dimension"
  const size_t ldb = k;
  const size_t ldc = m;
  const dtype alpha = 0.1;
  const dtype beta = 0.0;

  std::cout << "Reading A..." << std::endl;
  std::vector<dtype> h_A = readVector<dtype>("./a.in", m * k);
  std::cout << "Done\nReading B... " << std::endl;
  std::vector<dtype> h_B = readVector<dtype>("./b.in", k * n);
  std::cout << "Done" << std::endl;
  std::vector<dtype> h_C(m * n);

  std::cout << h_A.size() << std::endl;
  std::cout << h_B.size() << std::endl;
  dtype *d_A = nullptr;
  dtype *d_B = nullptr;
  dtype *d_C = nullptr;

  printf("A\n");
  print_matrix(m, k, h_A.data(), lda);
  printf("=====\n");

  printf("B\n");
  print_matrix(k, n, h_B.data(), ldb);
  printf("=====\n");

  uint64_t t0 = get_timestamp_in_microsec();

  CUBLAS_CHECK(hipblasCreate(&cublasH));

  uint64_t t1 = get_timestamp_in_microsec();
  /* step 2: copy data to device */
  CUDA_CHECK(hipMalloc((void **)(&d_A), sizeof(dtype) * m * k));
  CUDA_CHECK(hipMalloc((void **)(&d_B), sizeof(dtype) * k * n));
  CUDA_CHECK(hipMalloc((void **)(&d_C), sizeof(dtype) * m * n));
  uint64_t t2 = get_timestamp_in_microsec();
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), sizeof(dtype) * m * k,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, h_B.data(), sizeof(dtype) * k * n,
                        hipMemcpyHostToDevice));

  hipDeviceSynchronize();
  uint64_t t3 = get_timestamp_in_microsec();
  /* When throwing error, the argument count starts from 0*/
  CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                           d_A, lda, d_B, ldb, &beta, d_C, ldc));
  hipDeviceSynchronize();
  uint64_t t4 = get_timestamp_in_microsec();

  CUDA_CHECK(hipMemcpy(h_C.data(), d_C, sizeof(dtype) * m * n,
                        hipMemcpyDeviceToHost));

  uint64_t t5 = get_timestamp_in_microsec();

  printf("C\n");
  print_matrix(m, n, h_C.data(), ldc);
  std::cout << "=====\nWriting C...\n";
  write_matrix_to_csv(h_C, m, n, "./cublas.csv.out");
  std::cout << "Done" << std::endl;

  /* free resources */
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUBLAS_CHECK(hipblasDestroy(cublasH));
  CUDA_CHECK(hipDeviceReset());
  std::cout << "CUDA Init: " << (t1 - t0) / 1000.0 << "ms\n"
            << "hipMalloc(): " << (t2 - t1) / 1000.0 << "ms\n"
            << "hipMemcpy(HostToDevice): " << (t3 - t2) / 1000.0 << "ms\n"
            << "hipblasDgemm(): " << (t4 - t3) / 1000.0 << "ms\n"
            << "hipMemcpy(DeviceToHost): " << (t5 - t4) / 1000.0 << "ms\n"
            << "Total: " << (t5 - t0) / 1000.0 << "ms" << std::endl;
  return EXIT_SUCCESS;
}
