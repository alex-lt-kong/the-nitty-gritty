// Originally
// https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuBLAS/Level-3/gemm
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

int main(int argc, char *argv[]) {
  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;

  const int m = 4;
  const int n = 3;
  const int k = n;
  int lda;
  int ldb;
  int ldc = n;
  /*
   *   A = | 1.0 | 2.0 |
   *       | 3.0 | 4.0 |
   *
   *   B = | 5.0 | 6.0 |
   *       | 7.0 | 8.0 |
   */

  double *A;
  generate_random_matrix(m, n, &A, &lda);
  double *B;
  generate_random_matrix(n, m, &B, &ldb);
  printf("lda: %d, ldb: %d\n", lda, ldb);
  double *C = (double *)malloc(sizeof(double) * m * m);
  const double alpha = 1.0;
  const double beta = 0.0;

  double *d_A = nullptr;
  double *d_B = nullptr;
  double *d_C = nullptr;

  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  printf("A\n");
  print_matrix(m, n, A, lda);
  printf("=====\n");

  printf("B\n");
  print_matrix(n, m, B, ldb);
  printf("=====\n");

  /* step 1: create cublas handle, bind a stream */
  CUBLAS_CHECK(hipblasCreate(&cublasH));

  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  /* step 2: copy data to device */
  CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * m * n));
  CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(double) * n * m));
  CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&d_C), sizeof(double) * m * m));

  CUDA_CHECK(hipMemcpyAsync(d_A, A, sizeof(double) * m * n,
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(d_B, B, sizeof(double) * n * m,
                             hipMemcpyHostToDevice, stream));

  /* step 3: compute */
  /* When throwing error, the argument count starts from 0*/
  CUBLAS_CHECK(hipblasDgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda,
                           d_B, ldb, &beta, d_C, m));

  /* step 4: copy data to host */
  CUDA_CHECK(hipMemcpyAsync(C, d_C, sizeof(double) * m * m,
                             hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));

  /*
   *   C = | 23.0 | 31.0 |
   *       | 34.0 | 46.0 |
   */

  printf("C\n");
  print_matrix(m, m, C, ldc);
  printf("=====\n");

  /* free resources */
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUBLAS_CHECK(hipblasDestroy(cublasH));

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipDeviceReset());

  return EXIT_SUCCESS;
}
