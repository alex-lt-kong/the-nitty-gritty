#include "hip/hip_runtime.h"
// Originally
// https://github.com/NVIDIA/CUDALibrarySamples/tree/master/cuBLAS/Level-3/gemm
// Doc: https://docs.nvidia.com/cuda/cublas/#cublas-t-gemm
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../../utils.h"
#include "../../utils.hpp"
#include "../cublas_utils.h"

using dtype = float;

__global__ void log_kernel(dtype *x, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = 0.1 * log(x[i] + 11.0);
  }
}

int main(int argc, char *argv[]) {

  hipblasHandle_t cublasH = NULL;
  hipStream_t stream = NULL;
  int device;
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDevice(&device));
  CUDA_CHECK(hipGetDeviceProperties(&prop, device));
  std::cout << "GPU: " << prop.name << std::endl;

  size_t m = 3000;
  size_t k = 1000;
  size_t n = 2000;
  const size_t lda = m; // ld means "leading dimension"
  const size_t ldb = k;
  const size_t ldc = m;
  const dtype alpha = 0.1;
  const dtype beta = 0.0;

  std::cout << "Reading A..." << std::endl;
  std::vector<dtype> h_A = readVector<dtype>("./a.in", m * k);
  std::cout << "Done (" << h_A.size() << ")\nReading B... " << std::endl;
  std::vector<dtype> h_B = readVector<dtype>("./b.in", k * n);
  std::cout << "Done (" << h_B.size() << ")" << std::endl;
  std::vector<dtype> h_C(m * n, 0.0);

  dtype *d_A = nullptr;
  dtype *d_B = nullptr;
  dtype *d_C = nullptr;

  printf("A\n");
  print_matrix(m, k, h_A.data(), lda);
  printf("=====\n");

  printf("B\n");
  print_matrix(k, n, h_B.data(), ldb);
  printf("=====\n");

  uint64_t t0 = get_timestamp_in_microsec();
  CUBLAS_CHECK(hipblasCreate(&cublasH));

  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

  CUDA_CHECK(hipMalloc((void **)(&d_A), sizeof(dtype) * m * k));
  CUDA_CHECK(hipMalloc((void **)(&d_B), sizeof(dtype) * k * n));
  CUDA_CHECK(hipMalloc((void **)(&d_C), sizeof(dtype) * m * n));

  CUDA_CHECK(hipMemcpyAsync(d_A, h_A.data(), sizeof(dtype) * m * k,
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(d_B, h_B.data(), sizeof(dtype) * k * n,
                             hipMemcpyHostToDevice, stream));

  int block_size = 256;
  // Changing it doesn't appear to have a significant impact on the
  // performance of log_kernel<<<num_blocks, block_size>>>(d_A, m * k)--it
  // always takes ~0.15ms
  int num_blocks = (m * k + block_size - 1) / block_size;

  log_kernel<<<num_blocks, block_size>>>(d_A, m * k);

  /* When throwing error, the argument count starts from 0*/
  CUBLAS_CHECK(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                           d_A, lda, d_B, ldb, &beta, d_C, ldc));

  CUDA_CHECK(hipMemcpyAsync(h_C.data(), d_C, sizeof(dtype) * m * n,
                             hipMemcpyDeviceToHost, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));
  uint64_t t1 = get_timestamp_in_microsec();
  std::cout << "C\n";
  print_matrix(m, n, h_C.data(), ldc);
  std::cout << "=====\nWriting C...\n";
  write_matrix_to_csv(h_C, m, n, "./cublas.csv.out");
  std::cout << "Done" << std::endl;

  /* free resources */
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));

  CUBLAS_CHECK(hipblasDestroy(cublasH));

  CUDA_CHECK(hipStreamDestroy(stream));

  CUDA_CHECK(hipDeviceReset());
  printf("%.02fms\n", (t1 - t0) / 1000.0);
  return EXIT_SUCCESS;
}
