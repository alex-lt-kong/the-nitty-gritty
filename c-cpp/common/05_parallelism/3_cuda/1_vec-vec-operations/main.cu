#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#include "../../utils.h"

typedef void calculationRoutine(const float *a, const float *b, float *c,
                                const ssize_t len);

/* The function name/signature doesn't matter, it will be transparently sent to
 * GPU to execute.
 * This function, to be executed on NVIDIA GPU, is also known as "CUDA kernel".
 */
__global__ void gpuVectorAdd(const float *a, const float *b, float *c,
                             const ssize_t len) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len) {
    c[i] = a[i] + b[i];
  }
}

__global__ void gpuVectorMul(const float *a, const float *b, float *c,
                             const ssize_t len) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len) {
    c[i] = a[i] * b[i];
  }
}

__global__ void gpuVectorDiv(const float *a, const float *b, float *c,
                             const ssize_t len) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len) {
    c[i] = a[i] / b[i];
  }
}

__global__ void gpuVectorPow(const float *a, const float *b, float *c,
                             const ssize_t len) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < len) {
    c[i] = pow(a[i], b[i]);
  }
}

void cpuVectorAdd(const float *a, const float *b, float *c, const ssize_t len) {
  for (int i = 0; i < len; ++i) {
    c[i] = a[i] + b[i];
  }
}

void cpuVectorMul(const float *a, const float *b, float *c, const ssize_t len) {
  for (int i = 0; i < len; ++i) {
    c[i] = a[i] * b[i];
  }
}

void cpuVectorDiv(const float *a, const float *b, float *c, const ssize_t len) {
  for (int i = 0; i < len; ++i) {
    c[i] = a[i] / b[i];
  }
}

void cpuVectorPow(const float *a, const float *b, float *c, const ssize_t len) {
  for (int i = 0; i < len; ++i) {
    c[i] = pow(a[i], b[i]);
  }
}

void callCPURoutine(calculationRoutine funcPtr, float *a, float *b, float *c,
                    const ssize_t len) {
  uint64_t t0, t1;
  uint64_t cpu_elapsed;
  printf("--- Running on CPU ---\n");
  t0 = get_timestamp_in_microsec();
  funcPtr(a, b, c, len);
  t1 = get_timestamp_in_microsec();
  cpu_elapsed = t1 - t0;
  printf("Done, took %.2lfms\n", cpu_elapsed / 1000.0);
}

void callGPURoutine(calculationRoutine funcPtr, float *a, float *b, float *c,
                    const ssize_t len) {

  hipError_t hipError_t;
  uint64_t t0, t1, elapsed, T0, T1;

  printf("--- Running on GPU ---\n");
  T0 = get_timestamp_in_microsec();
  float *cudaA = NULL;
  float *cudaB = NULL;
  float *cudaC = NULL;
  int threadsPerBlock, blocksPerGrid;
  // Allocate memory for pointers into the GPU
  if ((hipError_t = hipMalloc(&cudaA, sizeof(float) * len)) != hipSuccess ||
      (hipError_t = hipMalloc(&cudaB, sizeof(float) * len)) != hipSuccess ||
      (hipError_t = hipMalloc(&cudaC, sizeof(float) * len)) != hipSuccess) {
    // C/C++ implements short-circuit evaluation, meaning that for the ||
    // operator, if the first argument is evaluted to true, the 2nd
    // argument will not be evaluted
    fprintf(stderr, "hipMalloc() failed: %s\n", hipGetErrorString(hipError_t));
    goto err_cuda_malloc;
  }

  t0 = get_timestamp_in_microsec();
  // Copy vectors into the GPU
  if ((hipError_t = hipMemcpy(cudaA, a, len * sizeof(float),
                              hipMemcpyHostToDevice)) != hipSuccess ||
      (hipError_t = hipMemcpy(cudaB, b, len * sizeof(float),
                              hipMemcpyHostToDevice)) != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed: %s\n", hipGetErrorString(hipError_t));
    goto err_cuda_memcpy;
  }
  hipDeviceSynchronize();
  t1 = get_timestamp_in_microsec();
  elapsed = t1 - t0;
  printf("Took %.2lfms to move data from RAM to GPU memory (%.1lfMB/sec)\n",
         elapsed / 1000.0,
         2 * len * sizeof(float) / 1024.0 / 1024 / (elapsed / 1000.0 / 1000.0));

  threadsPerBlock = 128;
  blocksPerGrid = (len + threadsPerBlock - 1) / threadsPerBlock;
  t0 = get_timestamp_in_microsec();
  funcPtr<<<blocksPerGrid, threadsPerBlock>>>(cudaA, cudaB, cudaC, len);
  hipDeviceSynchronize();
  t1 = get_timestamp_in_microsec();
  printf("Took %.2lfms to calculate\n", (t1 - t0) / 1000.0);

  t0 = get_timestamp_in_microsec();
  if ((hipError_t = hipMemcpy(c, cudaC, len * sizeof(float),
                              hipMemcpyDeviceToHost)) != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed: %s\n", hipGetErrorString(hipError_t));
    goto err_cuda_memcpy;
  }
  t1 = get_timestamp_in_microsec();
  elapsed = t1 - t0;
  printf("Took %.2lfms to move data from GPU memory to RAM (%.1lfMB/sec)\n",
         elapsed / 1000.0,
         len * sizeof(float) / 1024.0 / 1024 / (elapsed / 1000.0 / 1000.0));
err_cuda_memcpy:
err_cuda_malloc:
  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(cudaC);
  T1 = get_timestamp_in_microsec();
  printf("Done, took %.2lfms\n", (T1 - T0) / 1000.0);
}

void prepareRandomNumbers(float *a, float *b, ssize_t len) {
  for (int i = 0; i < len; ++i) {
    a[i] = rand() % (RAND_MAX / 2 - 1);
    b[i] = rand() % (RAND_MAX / 2 - 1);
    a[i] /= (float)RAND_MAX;
    b[i] /= (float)RAND_MAX;
    if (fabs(a[i]) < 0.01) {
      a[i] += 1;
    }
    if (fabs(b[i]) < 0.01) {
      b[i] += 1;
    }
  }
  printf("%.1lf MB random data generated\n",
         2 * len * sizeof(float) / 1024.0 / 1024);
}

void checkResults(const float *c_cpu, const float *c_gpu, const ssize_t len) {
  printf("\nChecking if CPU/GPU results are identical...");

  int inconsistent_count = 0;
  for (int i = 0; i < len; ++i) {
    if (fabs(c_cpu[i] - c_gpu[i]) > 1e-6) {
      fprintf(stderr, "%d-th element is DIFFERENT (%lf vs %lf)!!!\n", i,
              c_cpu[i], c_gpu[i]);
      ++inconsistent_count;
    }
    if (inconsistent_count >= 10) {
      fprintf(stderr, "too many mismatches, check aborted\n");
      break;
    }
  }
  if (inconsistent_count == 0) {
    printf("YES!\n\n");
  } else {
    fprintf(stderr, "%d mismatches found!\n\n", inconsistent_count);
  }
}

void printCPUandGPU() {
  FILE *fp = fopen("/proc/cpuinfo", "r");
  char line[PATH_MAX];
  char *version = NULL;

  while (fgets(line, PATH_MAX, fp)) {
    if (strstr(line, "model name") != NULL) {
      version = strchr(line, ':') + 2; // skip over ": "
      break;
    }
  }

  fclose(fp);

  if (version) {
    printf("CPU: %s", version);
  } else {
    printf("Failed to get CPU version.");
  }

  int device;
  hipDeviceProp_t prop;
  hipError_t hipError_t;
  if ((hipError_t = hipGetDevice(&device)) != hipSuccess ||
      (hipError_t = hipGetDeviceProperties(&prop, device)) != hipSuccess) {
    fprintf(stderr, "hipGetDevice() failed: %s\n",
            hipGetErrorString(hipError_t));
  }
  printf("GPU: %s\n\n", prop.name);
}

int main(void) {
  int retval = 0;
  printCPUandGPU();
  const ssize_t len = 200 * 1000 * 1000;
  float *a = (float *)malloc(len * sizeof(float));
  float *b = (float *)malloc(len * sizeof(float));
  float *cCPU = (float *)calloc(len, sizeof(float));
  float *cGPU = (float *)calloc(len, sizeof(float));

  srand(time(NULL));

  calculationRoutine *cpuFuncPtrs[] = {&cpuVectorAdd, &cpuVectorMul,
                                       &cpuVectorDiv, &cpuVectorPow};
  calculationRoutine *gpuFuncPtrs[] = {&gpuVectorAdd, &gpuVectorMul,
                                       &gpuVectorDiv, &gpuVectorPow};
  char routineNames[][32] = {"vectorAdd", "vectorMul", "vectorDiv",
                             "vectorPow"};

  if (a == NULL || b == NULL || cCPU == NULL || cGPU == NULL) {
    fprintf(stderr, "malloc() failed\n");
    goto err_malloc;
  }
  prepareRandomNumbers(a, b, len);

  for (int i = 0; i < sizeof(cpuFuncPtrs) / sizeof(calculationRoutine *); ++i) {
    printf("\n========== Now running: %s ==========\n", routineNames[i]);
    (void)callCPURoutine(cpuFuncPtrs[i], a, b, cCPU, len);
    (void)callGPURoutine(gpuFuncPtrs[i], a, b, cGPU, len);
    checkResults(cCPU, cGPU, len);
  }

err_malloc:
  free(a);
  free(b);
  free(cCPU);
  free(cGPU);
  return retval;
}